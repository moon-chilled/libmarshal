#include "hip/hip_runtime.h"
//===--- marshal.cu - GPU in-place marshaling library          ----------===//
// (C) Copyright 2012 The Board of Trustees of the University of Illinois.
// All rights reserved.
//
//                            libmarshal
// Developed by:
//                           IMPACT Research Group
//                  University of Illinois, Urbana-Champaign
// 
// This file is distributed under the Illinois Open Source License.
// See LICENSE.TXT for details.
//
// Author: I-Jui Sung (sung10@illinois.edu)
//
//===---------------------------------------------------------------------===//
//
//  This file defines the interface of the libmarshal 
//
//===---------------------------------------------------------------------===//

#include <cstdlib>
#include <cassert>
#include <iostream>
#include "marshal.h"
#include "marshal_kernel.cu"
extern "C" bool gpu_aos_asta_bs(float *src, int height, int width,
    int tile_size, clock_t *timer) {
  assert ((height/tile_size)*tile_size == height);
  dim3 threads (width, tile_size, 1);
  BS_marshal<<<height/tile_size, threads>>>(src, tile_size, width, timer);
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    std::cerr << hipGetErrorString(err) << std::endl;
  }
  return hipSuccess != err;
}

#define NR_THREADS 64
extern "C" bool gpu_aos_asta_pttwac(float *src, int height, int width,
    int tile_size, clock_t *timer) {
  assert ((height/tile_size)*tile_size == height);
  PTTWAC_marshal<<<min(height/tile_size,1024), NR_THREADS,
    ((tile_size*width+31)/32)*4>>>(src, height, tile_size, width, timer);
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    std::cerr << hipGetErrorString(err) << std::endl;
  }
  return hipSuccess != err;
}

extern "C" bool gpu_soa_asta_pttwac(float *src, int height, int width,
    int tile_size, clock_t *timer) {
  assert ((height/tile_size)*tile_size == height);
  int *finished;
  hipMalloc(&finished, height*width/tile_size*sizeof(int));
  hipMemset(finished, 0, height*width/tile_size*sizeof(int));

  size_t grid = min(height/tile_size*width, 1024);
  PTTWAC_marshal_soa<<<grid, tile_size>>>(
      src, height, tile_size, width, finished, timer);
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    std::cerr << hipGetErrorString(err) << std::endl;
  }
  hipFree(finished);
  return hipSuccess != err;
}
