#include <cstdlib>
#include <cassert>
#include <iostream>
#include "marshal.h"
#include "marshal_kernel.cu"
extern "C" bool gpu_aos_asta_bs(float *src, int height, int width,
    int tile_size, clock_t *timer) {
  assert ((height/tile_size)*tile_size == height);
  dim3 threads (width, tile_size, 1);
  BS_marshal<<<height/tile_size, threads>>>(src, tile_size, width, timer);
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    std::cerr << hipGetErrorString(err) << std::endl;
  }
  return hipSuccess != err;
}

extern "C" bool gpu_aos_asta_pttwac(float *src, int height, int width,
    int tile_size, clock_t *timer) {
  assert ((height/tile_size)*tile_size == height);
  PTTWAC_marshal<<<height/tile_size, 16>>>(src, tile_size, width, timer);
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    std::cerr << hipGetErrorString(err) << std::endl;
  }
  return hipSuccess != err;
}
